#include "physics.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"


int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

	int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
	compute_forces_gpu <<< blks, NUM_THREADS >>> (d_particles, n);
        
        //
        //  move particles
        //
	move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
